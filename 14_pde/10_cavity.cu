
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <array>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cassert>

constexpr int nx = 41;
constexpr int ny = 41;
constexpr int nt = 500;
constexpr int nit = 50;

constexpr double range_l = 0;
constexpr double range_r = 2;
constexpr double dx = (range_r - range_l) / static_cast<double>(nx - 1);
constexpr double dy = (range_r - range_l) / static_cast<double>(ny - 1);
constexpr double dt = 0.01;
constexpr double rho = 1;
constexpr double nu = 0.02;

template<typename T>
__host__ __device__ inline constexpr T pow2(const T& v) {
	return v * v;
}


using MATRIX = std::array<std::array<double, nx>, ny>;
std::ofstream pyplot_out{ "pyplot.py" };
void pyplot_init() {
	pyplot_out << R"(
import numpy as np
import matplotlib.pyplot as plt

x = np.linspace(0, 2, )" << nx << R"()
y = np.linspace(0, 2, )" << ny << R"()
X, Y = np.meshgrid(x, y)

)";
}
void pyplot_array(const MATRIX& m) {
	auto convert = [](double d)->double {
		if (std::isfinite(d)) { return d; }
		if (std::signbit(d)) { return -1e18; }
		return 1e18;
	};
	pyplot_out << "np.array([";
	for (int h = 0; h < ny; ++h) {
		pyplot_out << '[';
		for (auto& vv : m[h]) {
			pyplot_out
				<< convert(vv) << ',';
		}
		pyplot_out << "],";
	}
	pyplot_out << "])\n";
}
void pyplot(const MATRIX& u, const MATRIX& v, const MATRIX& p) {

	pyplot_out << "u=";
	pyplot_array(u);

	pyplot_out << "v=";
	pyplot_array(v);

	pyplot_out << "p=";
	pyplot_array(p);

	pyplot_out << R"(
plt.contourf(X, Y, p, alpha=0.5, cmap=plt.cm.coolwarm)
plt.quiver(X[::2, ::2], Y[::2, ::2], u[::2, ::2], v[::2, ::2])
plt.pause(.001)
plt.clf()
del u
del v
del p
)";
}

__global__ void step1(double* out_b, double* prev_u, double* prev_v) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const auto un = reinterpret_cast<const double(*)[nx]>(prev_u);
	const auto vn = reinterpret_cast<const double(*)[nx]>(prev_v);
	const auto b = reinterpret_cast<double(*)[nx]>(out_b);
	const int h = 1 + (idx / (nx - 2));
	const int w = 1 + (idx % (nx - 2));

	if (ny - 1 <= h) { return; }
	assert(1 <= h && h < ny - 1);
	assert(1 <= w && w < nx - 1);
	b[h][w] = rho * (1 / dt * \
		((un[h][w + 1] - un[h][w - 1]) / (2 * dx) + (vn[h + 1][w] - vn[h - 1][w]) / (2 * dy)) -
		pow2((un[h][w + 1] - un[h][w - 1]) / (2 * dx)) - 2 * ((un[h + 1][w] - un[h - 1][w]) / (2 * dy) *
			(vn[h][w + 1] - vn[h][w - 1]) / (2 * dx)) - pow2((vn[h + 1][w] - vn[h - 1][w]) / (2 * dy)));
}

__global__ void step2(double* out_p, double* prev_p, double* now_b) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const auto p = reinterpret_cast<double(*)[nx]>(out_p);
	const auto pn = reinterpret_cast<const double(*)[nx]>(prev_p);
	const auto b = reinterpret_cast<const double(*)[nx]>(now_b);
	int h = (idx / nx);
	int w = (idx % nx);

	if (ny <= h) { return; }
	assert(0 <= h && h < ny);
	assert(0 <= w && w < nx);
	auto& out = p[h][w];
	if (w == nx - 1) { w = nx - 2; }
	else if (w == 0) { w = 1; }
	if (h == 0) { h = 1; }

	if (h == ny - 1) {
		out = 0;
	}
	else {
		assert(1 <= h && h < ny - 1);
		assert(1 <= w && w < nx - 1);
		out = (
			pow2(dy) * (pn[h][w + 1] + pn[h][w - 1]) +
			pow2(dx) * (pn[h + 1][w] + pn[h - 1][w]) -
			b[h][w] * pow2(dx * dy)
			) / (2 * (pow2(dx) + pow2(dy)));
	}
}

__global__ void step3(double* out_u, double* out_v, double* prev_u, double* prev_v, double* now_p) {
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const auto u = reinterpret_cast<double(*)[nx]>(out_u);
	const auto v = reinterpret_cast<double(*)[nx]>(out_v);
	const auto un = reinterpret_cast<const double(*)[nx]>(prev_u);
	const auto vn = reinterpret_cast<const double(*)[nx]>(prev_v);
	const auto p = reinterpret_cast<const double(*)[nx]>(now_p);
	int h = (idx / nx);
	int w = (idx % nx);

	if (ny <= h) { return; }
	assert(0 <= h && h < ny);
	assert(0 <= w && w < nx);
	if (h == ny - 1) {
		u[h][w] = 1;
		v[h][w] = 0;
	}
	else if (h == 0 || w == 0 || w == nx - 1) {
		u[h][w] = 0;
		v[h][w] = 0;
	}
	else {
		assert(1 <= h && h < ny - 1);
		assert(1 <= w && w < nx - 1);
		u[h][w] = un[h][w]
			- un[h][w] * dt / dx * (un[h][w] - un[h][w - 1])
			- un[h][w] * dt / dy * (un[h][w] - un[h - 1][w])
			- dt / (2 * rho * dx) * (p[h][w + 1] - p[h][w - 1])
			+ nu * dt / pow2(dx) * (un[h][w + 1] - 2 * un[h][w] + un[h][w - 1])
			+ nu * dt / pow2(dy) * (un[h + 1][w] - 2 * un[h][w] + un[h - 1][w]);
		v[h][w] = vn[h][w]
			- vn[h][w] * dt / dx * (vn[h][w] - vn[h][w - 1])
			- vn[h][w] * dt / dy * (vn[h][w] - vn[h - 1][w])
			- dt / (2 * rho * dx) * (p[h + 1][w] - p[h - 1][w])
			+ nu * dt / pow2(dx) * (vn[h][w + 1] - 2 * vn[h][w] + vn[h][w - 1])
			+ nu * dt / pow2(dy) * (vn[h + 1][w] - 2 * vn[h][w] + vn[h - 1][w]);
	}
}

void run()
{
	//init pyplot
	pyplot_init();

	//static std::array<double, nx> x;
	//for (int i = 0; i < nx; ++i) { x[i] = dx * i; }
	//static std::array<double, ny> y;
	//for (int i = 0; i < ny; ++i) { y[i] = dy * i; }

	static std::array<std::array<double, nx>, ny> host_u, host_v, host_p;
	double* u_;
	double* v_;
	double* p_;
	double* b;
	hipMalloc((void**)&u_, 2*nx*ny*sizeof(double)); hipMemsetAsync(u_, 0, 2*nx*ny*sizeof(double));
	hipMalloc((void**)&v_, 2*nx*ny*sizeof(double)); hipMemsetAsync(v_, 0, 2*nx*ny*sizeof(double));
	hipMalloc((void**)&p_, 2*nx*ny*sizeof(double)); hipMemsetAsync(p_, 0, 2*nx*ny*sizeof(double));
	hipMalloc((void**)&b, nx*ny*sizeof(double));
	for (int n = 0; n < nt; ++n) {
		//un = u.copy();
		//vn = v.copy();
		auto u = u_ + ((n % 2 == 0) ? 0 : nx * ny);
		auto un = u_ + ((n % 2 != 0) ? 0 : nx * ny);
		auto v = v_ + ((n % 2 == 0) ? 0 : nx * ny);
		auto vn = v_ + ((n % 2 != 0) ? 0 : nx * ny);

		constexpr int block_num = (nx*ny + 1023) / 1024;
		step1<<<block_num, 1024>>>(b,un,vn);
		for (int it = 0; it < nit; ++it) {
			auto p = p_ + (((n + it) % 2 == 0) ? 0 : nx * ny);
			auto pn = p_ + (((n + it) % 2 != 0) ? 0 : nx * ny);
			step2<<<block_num, 1024>>>(p, pn, b);
		}
		auto p = p_ + (((n + nit) % 2 != 0) ? 0 : nx * ny);
		step3<<<block_num, 1024>>>(u, v, un, vn, p);

		hipMemcpyAsync(host_u[0].data(), u, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpyAsync(host_v[0].data(), v, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpyAsync(host_p[0].data(), p, nx*ny*sizeof(double), hipMemcpyDeviceToHost);
		pyplot_out << "plt.title(\"" << n << "\")\n";
		hipDeviceSynchronize();
		pyplot(host_u, host_v, host_p);
	}

	//finalize pyplot
	pyplot_out << "plt.show()" << std::endl;
}

int main()
{
	run();
	system("python3 pyplot.py");
	return 0;
}
