
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <array>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <cassert>

constexpr int nx = 41;
constexpr int ny = 41;
constexpr int nt = 500;
constexpr int nit = 50;

constexpr double range_l = 0;
constexpr double range_r = 2;
constexpr double dx = (range_r - range_l) / static_cast<double>(nx - 1);
constexpr double dy = (range_r - range_l) / static_cast<double>(ny - 1);
constexpr double dt = 0.01;
constexpr double rho = 1;
constexpr double nu = 0.02;

template<typename T>
inline constexpr T pow2(const T& v) {
	return v * v;
}


using MATRIX = std::array<std::array<double, nx>, ny>;
std::ofstream pyplot_out{ "pyplot.py" };
void pyplot_init() {
	pyplot_out << R"(
import numpy as np
import matplotlib.pyplot as plt

x = np.linspace(0, 2, )" << nx << R"()
y = np.linspace(0, 2, )" << ny << R"()
X, Y = np.meshgrid(x, y)

)";
}
void pyplot_array(const MATRIX& m) {
	auto convert = [](double d)->double {
		if (std::isfinite(d)) { return d; }
		if (std::signbit(d)) { return -1e18; }
		return 1e18;
	};
	pyplot_out << "np.array([";
	for (int h = 0; h < ny; ++h) {
		pyplot_out << '[';
		for (auto& vv : m[h]) {
			pyplot_out
				<< convert(vv) << ',';
		}
		pyplot_out << "],";
	}
	pyplot_out << "])\n";
}
void pyplot(const MATRIX& u, const MATRIX& v, const MATRIX& p) {

	pyplot_out << "u=";
	pyplot_array(u);

	pyplot_out << "v=";
	pyplot_array(v);

	pyplot_out << "p=";
	pyplot_array(p);

	pyplot_out << R"(
plt.contourf(X, Y, p, alpha=0.5, cmap=plt.cm.coolwarm)
plt.quiver(X[::2, ::2], Y[::2, ::2], u[::2, ::2], v[::2, ::2])
plt.pause(.001)
plt.clf()
del u
del v
del p
)";
}

void step1(double* out_b, double* prev_u, double* prev_v, int idx) {
	//const int idx = blockIdx.x * blockDim_x + threadIdx.x;
	const auto un = reinterpret_cast<const double(*)[nx]>(prev_u);
	const auto vn = reinterpret_cast<const double(*)[nx]>(prev_v);
	const auto b = reinterpret_cast<double(*)[nx]>(out_b);
	const int h = 1 + (idx / (nx - 2));
	const int w = 1 + (idx % (nx - 2));

	assert(1 <= h && h < ny - 1);
	assert(1 <= w && w < nx - 1);
	if (ny - 1 <= h) { return; }
	b[h][w] = rho * (1 / dt * \
		((un[h][w + 1] - un[h][w - 1]) / (2 * dx) + (vn[h + 1][w] - vn[h - 1][w]) / (2 * dy)) -
		pow2((un[h][w + 1] - un[h][w - 1]) / (2 * dx)) - 2 * ((un[h + 1][w] - un[h - 1][w]) / (2 * dy) *
			(vn[h][w + 1] - vn[h][w - 1]) / (2 * dx)) - pow2((vn[h + 1][w] - vn[h - 1][w]) / (2 * dy)));
}

void step2(double* out_p, double* prev_p, double* now_b, int idx) {
	//const int idx = blockIdx.x * blockDim_x + threadIdx.x;
	const auto p = reinterpret_cast<double(*)[nx]>(out_p);
	const auto pn = reinterpret_cast<const double(*)[nx]>(prev_p);
	const auto b = reinterpret_cast<const double(*)[nx]>(now_b);
	int h = (idx / nx);
	int w = (idx % nx);

	assert(0 <= h && h < ny);
	assert(0 <= w && w < nx);
	if (ny <= h) { return; }
	auto& out = p[h][w];
	if (w == nx - 1) { w = nx - 2; }
	else if (w == 0) { w = 1; }
	if (h == 0) { h = 1; }

	if (h == ny - 1) {
		out = 0;
	}
	else {
		assert(1 <= h && h < ny - 1);
		assert(1 <= w && w < nx - 1);
		out = (
			pow2(dy) * (pn[h][w + 1] + pn[h][w - 1]) +
			pow2(dx) * (pn[h + 1][w] + pn[h - 1][w]) -
			b[h][w] * pow2(dx * dy)
			) / (2 * (pow2(dx) + pow2(dy)));
	}
}

void step3(double* out_u, double* out_v, double* prev_u, double* prev_v, double* now_p, int idx) {
	//const int idx = blockIdx.x * blockDim_x + threadIdx.x;
	const auto u = reinterpret_cast<double(*)[nx]>(out_u);
	const auto v = reinterpret_cast<double(*)[nx]>(out_v);
	const auto un = reinterpret_cast<const double(*)[nx]>(prev_u);
	const auto vn = reinterpret_cast<const double(*)[nx]>(prev_v);
	const auto p = reinterpret_cast<const double(*)[nx]>(now_p);
	int h = (idx / nx);
	int w = (idx % nx);

	assert(0 <= h && h < ny);
	assert(0 <= w && w < nx);
	if (ny <= h) { return; }
	if (h == ny - 1) {
		u[h][w] = 1;
		v[h][w] = 0;
	}
	else if (h == 0 || w == 0 || w == nx - 1) {
		u[h][w] = 0;
		v[h][w] = 0;
	}
	else {
		assert(1 <= h && h < ny - 1);
		assert(1 <= w && w < nx - 1);
		u[h][w] = un[h][w]
			- un[h][w] * dt / dx * (un[h][w] - un[h][w - 1])
			- un[h][w] * dt / dy * (un[h][w] - un[h - 1][w])
			- dt / (2 * rho * dx) * (p[h][w + 1] - p[h][w - 1])
			+ nu * dt / pow2(dx) * (un[h][w + 1] - 2 * un[h][w] + un[h][w - 1])
			+ nu * dt / pow2(dy) * (un[h + 1][w] - 2 * un[h][w] + un[h - 1][w]);
		v[h][w] = vn[h][w]
			- vn[h][w] * dt / dx * (vn[h][w] - vn[h][w - 1])
			- vn[h][w] * dt / dy * (vn[h][w] - vn[h - 1][w])
			- dt / (2 * rho * dx) * (p[h + 1][w] - p[h - 1][w])
			+ nu * dt / pow2(dx) * (vn[h][w + 1] - 2 * vn[h][w] + vn[h][w - 1])
			+ nu * dt / pow2(dy) * (vn[h + 1][w] - 2 * vn[h][w] + vn[h - 1][w]);
	}
}

void run()
{
	//init pyplot
	pyplot_init();

	//static std::array<double, nx> x;
	//for (int i = 0; i < nx; ++i) { x[i] = dx * i; }
	//static std::array<double, ny> y;
	//for (int i = 0; i < ny; ++i) { y[i] = dy * i; }

	static std::array<std::array<double, nx>, ny> u_[2], v_[2], p_[2], b;
	for (int n = 0; n < nt; ++n) {
		//un = u.copy();
		//vn = v.copy();
		auto& u = u_[n % 2];
		auto& un = u_[(n % 2) ^ 1];
		auto& v = v_[n % 2];
		auto& vn = v_[(n % 2) ^ 1];

		for (int h = 1; h < ny - 1; ++h) {
			for (int w = 1; w < nx - 1; ++w) {
				step1(b.data()->data(), un.data()->data(), vn.data()->data(), (h - 1) * (nx - 2) + (w - 1));
			}
		}
		//
		for (int it = 0; it < nit; ++it) {
			auto& p = p_[(n + it) % 2];
			auto& pn = p_[((n + it) % 2) ^ 1];
			//pn = p.copy();
			for (int h = 0; h < ny; ++h) {
				for (int w = 0; w < nx; ++w) {
					step2(p.data()->data(), pn.data()->data(), b.data()->data(), h * nx + w);
				}
			}
		}
		auto& p = p_[((n + nit) % 2) ^ 1];
		for (int h = 0; h < ny; ++h) {
			for (int w = 0; w < nx; ++w) {
				step3(u.data()->data(), v.data()->data(), un.data()->data(), vn.data()->data(), p.data()->data(), h * nx + w);
			}
		}

		pyplot_out << "plt.title(\"" << n << "\")\n";
		pyplot(u, v, p);
	}

	//finalize pyplot
	pyplot_out << "plt.show()" << std::endl;
}

int main()
{
	run();
	system("python3 pyplot.py");
	return 0;
}
